#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "matrix.hpp"

void host_error_and_exit(const char* error_msg) {
  perror(error_msg);
  // NOLINTNEXTLINE(concurrency-mt-unsafe)s
  exit(EXIT_FAILURE);
}

void check_device_error(const char* error_msg, hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s: %s\n", error_msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

Matrix* create_matrix_host(int num_rows, int num_cols) {
  Matrix* mat = (Matrix*)malloc(sizeof(Matrix));

  if (mat == NULL) {
    host_error_and_exit("Allocating host matrix struct");
  }

  mat->rows = num_rows;
  mat->cols = num_cols;

  mat->elements = (float*)malloc(sizeof(float) * (num_rows * num_cols));

  if (mat->elements == NULL) {
    host_error_and_exit("Allocating elements array");
  }
  return mat;
}

Matrix* create_matrix_device(int num_rows, int num_cols) {
  // First we create the memory for matrix struct on the device
  // with a pointer to it on host
  Matrix* d_mat;
  check_device_error("Allocating Matrix struct",
                     hipMalloc(&d_mat, sizeof(Matrix)));

  // Then create the memory for the elements on the device with
  // a pointer on host
  float* d_elems;
  check_device_error("Allocating elements array",
                     hipMalloc(&d_elems, sizeof(float) * num_rows * num_cols));

  // Then create the whole struct on host
  Matrix h_mat;
  h_mat.rows = num_rows;
  h_mat.cols = num_cols;
  h_mat.elements = d_elems;

  // Finally move the whole struct to device
  check_device_error(
      "Copying struct to device",
      hipMemcpy(d_mat, &h_mat, sizeof(Matrix), hipMemcpyHostToDevice));

  return d_mat;
}

void free_matrix_host(Matrix* h_mat) {
  free(h_mat->elements);
  free(h_mat);
}

void free_matrix_device(Matrix* d_mat) {
  // First, copy the struct back to host so we can read the pointers
  Matrix h_mat;
  check_device_error(
      "Copy struct to host",
      hipMemcpy(&h_mat, d_mat, sizeof(Matrix), hipMemcpyDeviceToHost));

  check_device_error("Free elements", hipFree(h_mat.elements));
  check_device_error("Free struct", hipFree(d_mat));
}

Matrix* copy_matrix_host_to_device(const Matrix* h_mat) {
  // First we create the memory for matrix struct on the device
  // with a pointer to it on host
  Matrix* d_mat;
  check_device_error("Allocating Matrix struct",
                     hipMalloc(&d_mat, sizeof(Matrix)));

  // Then create the memory for the elements on the device with
  // a pointer on host
  float* d_elems;
  check_device_error(
      "Allocating elements array",
      hipMalloc(&d_elems, sizeof(float) * h_mat->rows * h_mat->cols));

  // Then create the whole struct on host
  Matrix temp_h_mat;
  temp_h_mat.rows = h_mat->rows;
  temp_h_mat.cols = h_mat->cols;
  temp_h_mat.elements = d_elems;

  // Copy the whole struct to device
  check_device_error(
      "Copying struct to device",
      hipMemcpy(d_mat, &temp_h_mat, sizeof(Matrix), hipMemcpyHostToDevice));

  // Copy the elements to the elements
  check_device_error("Copy elements to device",
                     hipMemcpy(d_elems, h_mat->elements,
                                sizeof(float) * (h_mat->rows * h_mat->cols),
                                hipMemcpyHostToDevice));

  return d_mat;
}

Matrix* copy_matrix_device_to_host(const Matrix* d_mat) {
  Matrix* h_mat = (Matrix*)malloc(sizeof(Matrix));
  if (h_mat == NULL) {
    host_error_and_exit("Allocate memory for matrix struct on host");
  }
  check_device_error(
      "Copy struct to host",
      hipMemcpy(h_mat, d_mat, sizeof(Matrix), hipMemcpyDeviceToHost));

  float* elements = (float*)malloc(sizeof(float) * h_mat->rows * h_mat->cols);
  if (elements == NULL) {
    host_error_and_exit("Allocate memory for elements on host");
  }

  check_device_error("Copy elements to host",
                     hipMemcpy(elements, h_mat->elements,
                                sizeof(float) * (h_mat->rows * h_mat->cols),
                                hipMemcpyDeviceToHost));
  h_mat->elements = elements;
  return h_mat;
}

Matrix* create_matrix_from_csv(const char* filename) {}
